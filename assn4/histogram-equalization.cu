#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

#define BLOCK_NUM 256
#define THREAD_NUM 256
void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    int i;
    for ( i = 0; i < nbr_bin; i ++){
        hist_out[i] = 0; //construct an array with one entry for each color grey
    }

    for ( i = 0; i < img_size; i ++){
        hist_out[img_in[i]] ++; //fill array with counts of pixels of that color in image
    }
}

void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
                            int * hist_in, int img_size, int nbr_bin){
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while(min == 0){
        min = hist_in[i++]; //find the number of darkest pixels in the image
    }
    d = img_size - min;
    for(i = 0; i < nbr_bin; i ++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if(lut[i] < 0){
            lut[i] = 0;
        }
        
        
    }
    
    /* Get the result image */
    for(i = 0; i < img_size; i ++){
        if(lut[img_in[i]] > 255){
            img_out[i] = 255;
        }
        else{
            img_out[i] = (unsigned char)lut[img_in[i]];
        }
        
    }
}

__global__ void histogram_gpu(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    
    __shared__ int temp[256];

    temp[threadIdx.x] = 0;
    __syncthreads();

    int id =  blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x
    if (id >= img_size)
    {
        return;
    }

    while (id < img_size){
        atomicAdd(&temp[img_in[id]],1);
        i+= offset;
    }
    __syncthreads();

    //unsigned char value = img_in[id];

    //int bin = value % nbr_bin;
    atomicAdd(&(hist_out[threadIdx.x]), temp[threadIdx.x]);
}

void getHist(int * hist_out, unsigned char* img_in, int img_size, int nbr_bin){

    unsigned char * dArray;
    hipMalloc(&dArray, img_size);
    hipMemcpy(dArray, img_in, img_size,hipMemcpyHostToDevice);

    int * dHist;
    hipMalloc(&dHist, nbr_bin * sizeof(int));
    hipMemset(dHist,0,nbr_bin * sizeof(int));

    dim3 block(32);
    dim3 grid((img_size + block.x - 1)/block.x);

    histogram_gpu<<<grid,block>>>(dHist,dArray,img_size,nbr_bin);

    hipMemcpy(hist_out,dHist, nbr_bin * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dArray);
    hipFree(dHist);



}

__global__ void histogram_image_compile_gpu(unsigned char * img_out, unsigned char * img_in,
                            int * lut, int image_size, int nbr_bin) {
        __shared__ unsigned int memlut[255];
     
        for(int i = 0; i < 255; i++){
            memlut[i] = lut[i]; //don't know if pointer is correct but I want a local copy of lut
        }
        int chunk_size = blockIdx.x; //need code here, we need to split the image array into local parts to run high performance calcs on
        int offset = image_size/blockIdx.x; //when getting a chunk of the in image, or writing a chunk to the out image, offset+i should map to the correct location
        __shared__ unsigned int local_img[9999]; //create a local version of a segment of the image to work against so the whole image isn't stored in gpu memory per core
        for(int i = 0; i < chunk_size; i ++) {
           local_img[i] = img_in[offset+i];
        }
        __syncthreads();
        for(int i = 0; i < chunk_size; i++) {
           local_img[i] = lut[local_img[i]];
        }
        __syncthreads();
        for(int i = 0; i < chunk_size; i++) {
       img_out[offset+i] = local_img[i];
       }
}


__host__ static void histogram_equalization_gpu(unsigned char * img_out, unsigned char * img_in, 
                            int * hist_in, int img_size, int nbr_bin){
    /* Calculating the lut doesn't really make sense as a massively parallel thing, as it's only going through a maximum of 255 steps
    so lets only cudaize the result image formation step	*/
    unsigned int lut[nbr_bin]; //look up table, same size as hist
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while(min == 0){
        min = hist_in[i++]; //find the number of darkest pixels in the image

    }
	d = img_size - min;
    for(i = 0; i < nbr_bin; i ++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if(lut[i] < 0){
            lut[i] = 0;
        }
    }    
       

	
	/* Get the result image*/
	
	//histogram_image_compile_gpu(img_out, img_in, lut, img_size, nbr_bin);
    
}

